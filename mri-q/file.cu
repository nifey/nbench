/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/


#include <hip/hip_runtime.h>
#include <endian.h>
#include <stdlib.h>
#include <malloc.h>
#include <stdio.h>
#include <inttypes.h>

#if __BYTE_ORDER != __LITTLE_ENDIAN
# error "File I/O is not implemented for this system: wrong endianness."
#endif

extern "C"
void inputData(char* fName, int* _numK, int* _numX,
               float** kx, float** ky, float** kz,
               float** x, float** y, float** z,
               float** phiR, float** phiI)
{
  int numK, numX;
  FILE* fid = fopen(fName, "r");

  if (fid == NULL)
    {
      fprintf(stderr, "Cannot open input file\n");
      exit(-1);
    }
  fread (&numK, sizeof (int), 1, fid);
  *_numK = numK;
  fread (&numX, sizeof (int), 1, fid);
  *_numX = numX;
  hipMallocManaged(kx, numK * sizeof (float));
  fread (*kx, sizeof (float), numK, fid);
  hipMallocManaged(ky, numK * sizeof (float));
  fread (*ky, sizeof (float), numK, fid);
  hipMallocManaged(kz, numK * sizeof (float));
  fread (*kz, sizeof (float), numK, fid);
  hipMallocManaged(x, numX * sizeof (float));
  fread (*x, sizeof (float), numX, fid);
  hipMallocManaged(y, numX * sizeof (float));
  fread (*y, sizeof (float), numX, fid);
  hipMallocManaged(z, numX * sizeof (float));
  fread (*z, sizeof (float), numX, fid);
  hipMallocManaged(phiR, numK * sizeof (float));
  fread (*phiR, sizeof (float), numK, fid);
  hipMallocManaged(phiI, numK * sizeof (float));
  fread (*phiI, sizeof (float), numK, fid);
  fclose (fid); 
}

extern "C"
void outputData(char* fName, float* outR, float* outI, int numX)
{
  FILE* fid = fopen(fName, "w");
  uint32_t tmp32;

  if (fid == NULL)
    {
      fprintf(stderr, "Cannot open output file\n");
      exit(-1);
    }

  /* Write the data size */
  tmp32 = numX;
  fwrite(&tmp32, sizeof(uint32_t), 1, fid);

  /* Write the reconstructed data */
  fwrite (outR, sizeof (float), numX, fid);
  fwrite (outI, sizeof (float), numX, fid);
  fclose (fid);
}
