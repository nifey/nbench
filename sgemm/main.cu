/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * Main entry of dense matrix-matrix multiplication kernel
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <parboil.h>
#include <iostream>
#include "sgemm_kernel.cu"

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, float **v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, float*);

extern "C"
void computeGold(float *, const float*, const float*, unsigned int, unsigned int, unsigned int);

int
main (int argc, char *argv[]) {

  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  int matArow, matAcol;
  int matBrow, matBcol;
  float *matA, *matBT;

  pb_InitializeTimerSet(&timers);

  /* Read command line. Expect 3 inputs: A, B and B^T 
     in column-major layout*/
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) 
      || (params->inpFiles[1] == NULL)
      || (params->inpFiles[2] == NULL)
      || (params->inpFiles[3] != NULL))
    {
      fprintf(stderr, "Expecting three input filenames\n");
      exit(-1);
    }
 
  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);

  // load A
  readColMajorMatrixFile(params->inpFiles[0],
      matArow, matAcol, &matA);

  // load B^T
  readColMajorMatrixFile(params->inpFiles[2],
      matBcol, matBrow, &matBT);

  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

  // CUDA memory allocation
  float *matC;
  hipMallocManaged(&matC, matArow * matBcol * sizeof(float));

  pb_SwitchToTimer( &timers, pb_TimerID_KERNEL );

  // Use standard sgemm interface
  regtileSgemm('N', 'T', matArow, matBcol, matAcol, 1.0f, \
      matA, matArow, matBT, matBcol, 0.0f, matC, matArow);

  hipDeviceSynchronize();
  if (params->outFile) {
    /* Write C to file */
    pb_SwitchToTimer(&timers, pb_TimerID_IO);
    writeColMajorMatrixFile(params->outFile,
	matArow, matBcol, matC); 
  }

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  double GPUtime = pb_GetElapsedTime(&(timers.timers[pb_TimerID_KERNEL]));
  std::cout<< "GFLOPs = " << 2.* matArow * matBcol * matAcol/GPUtime/1e9 << std::endl;
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);
  hipFree(matA);
  hipFree(matBT);
  hipFree(matC);
  return 0;
}
