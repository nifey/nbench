/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* I/O routines for reading and writing matrices in column-major
 * layout
 */


#include <hip/hip_runtime.h>
#include<fstream>
#include<iostream>
#include<vector>

bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, float **v)
{
  std::cerr << "Opening file:"<< fn << std::endl;
  std::fstream f(fn, std::fstream::in);
  if ( !f.good() ) {
    return false;
  }

  // Read # of rows and cols
  f >> nr_row;
  f >> nr_col;

  hipMallocManaged(v, nr_row * nr_col * sizeof(float));
  std::cerr << "Matrix dimension: "<<nr_row<<"x"<<nr_col<<std::endl;
  float data;
  unsigned k = 0;
  while (f.good() ) {
    f >> data;
    if (k < nr_row * nr_col) {
	*(*v + k) =  data;
	k++;
    }
  }

}

bool writeColMajorMatrixFile(const char *fn, int nr_row, int nr_col, float *v)
{
  std::cerr << "Opening file:"<< fn << " for write." << std::endl;
  std::fstream f(fn, std::fstream::out);
  if ( !f.good() ) {
    return false;
  }

  // Read # of rows and cols
  f << nr_row << " "<<nr_col<<" ";

  float data;
  std::cerr << "Matrix dimension: "<<nr_row<<"x"<<nr_col<<std::endl;
  for (int i = 0; i < nr_row * nr_col; ++i) {
    f << v[i] << ' ';
  }
  f << "\n";
  return true;

}
